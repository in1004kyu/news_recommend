#include "hip/hip_runtime.h"
#include<thrust/version.h>
#include<iostream>
#include<thrust/device_vector.h>

#include<stdlib.h>
#include<time.h>
#include<stdio.h>

#include"nvmuldiv.h"
#define MAX_BLOCKS 65535
#define MAX_THREADS_PER_BLOCK 1024

#define ENABLE_WARP_SCHEDULING


__global__ void kern_reduce_divmul (const float *v1,const float *v2,float d_v[],float d_r[],int len,int len_r)
{
	extern __shared__ float sdata[];

	unsigned int tid = threadIdx.x; //0 ..127
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
	sdata[tid] = (v1[i]/v2[i] ) * (v1[i+blockDim.x]/v2[i+blockDim.x]);
	__syncthreads();

#ifdef ENABLE_WARP_SCHEDULING
	for(unsigned int s=blockDim.x/2; s>32; s>>=1){
		if(tid < s) {
			sdata[tid] *= sdata[tid+s];
		}
		__syncthreads();
	}
	if( tid < 32 ){
		volatile float *smem =sdata;
		smem[tid] *= smem[tid+32];
		smem[tid] *= smem[tid+16];
		smem[tid] *= smem[tid+8];
		smem[tid] *= smem[tid+4];
		smem[tid] *= smem[tid+2];
		smem[tid] *= smem[tid+1];
	}
#else
	for(unsigned int s=blockDim.x/2; s>0; s>>=1){
		if( tid < s) {
			sdata[tid] *= sdata[tid+s];
		}
		__syncthreads();
	}
#endif
	if( tid == 0 ){
		d_r[blockIdx.x] = sdata[0];
	}
}

extern "C" int reduce_divmul_seg_devmem( float *v1, float *v2, float d_v[] , int len, float d_r[] ,int len_r)
{
	hipError_t err = hipSuccess;

	int numSegs = len / len_r;
	if( numSegs > MAX_BLOCKS ){
		fprintf(stderr, "reduce_divmul_seg: too many segments (%d / %d = %d > MAX_BLOCKS (%d) )\n",len , len_r, len/len_r,MAX_BLOCKS);
		return -1;
	}
	int dimGrid = numSegs;
	int aligned_len_r = len_r;
	int padded_len = dimGrid * aligned_len_r;
	int dimBlock = aligned_len_r / 2;
	size_t smemSize = aligned_len_r/2 * sizeof(float);

	if( padded_len != len){
		fprintf(stderr, "len(%d)/len_r(%d) != padded_len(%d)\n", len,len_r,padded_len);
		return -2;
	}
	
	kern_reduce_divmul<<< dimGrid, dimBlock, smemSize >>> (v1,v2,d_v,d_r,len,len_r);
	err = hipGetLastError();

	if(err != hipSuccess){
		fprintf(stderr, "CUDA (error code : %x: %s)\n", err, hipGetErrorString(err));
	}

	return 0;
}
