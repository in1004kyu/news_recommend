
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

__global__ void
vectorDiv(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] / B[i];
    }
}

int vdiv_cuda( float v1[], float v2[], float vr[], int len )
{
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = len;
    size_t size = numElements * sizeof(float);
    //printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    bool bPageLocked = true;
#if 0
    // Try if vr is page locked mermory
    err = cudaHostGetDevicePointer( &d_C, vr, 0 );
    if ( err != cudaSuccess ) {
        d_C = NULL;
        err = cudaMalloc((void **)&d_C, size);
    bPageLocked = false;
        fprintf( stderr, "Use of device memory for result at %p\n", d_C );
    } else {
        fprintf( stderr, "Use of page locked memory at %p\n", vr );
    }
#else
        err = hipMalloc((void **)&d_C, size);
    bPageLocked = false;
#endif


    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
//    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, v1, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, v2, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("vdiv:CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorDiv<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorDiv kernel (error code %x:%s)!\n", err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
//    printf("Copy output data from the CUDA device to the host memory\n");
    if (!bPageLocked) {
        err = hipMemcpy(vr, d_C, size, hipMemcpyDeviceToHost);

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }


#ifdef __NVMULDIV_ENABLE_VERIFY__
#if 0
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(v1[i] / v2[i] - vr[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
        fprintf(stderr, "v1[i]: %f v2[i]: %f vr[i]:%f, expected:%f\n", v1[i], v2[i], vr[i], v1[i]/v2[i]);
        }
    }
#endif
#endif

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    if ( !bPageLocked) {
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    }

    // Reset the device and exit
#if 1
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
#endif

//    printf("Done\n");
    return 0;
}

int vdiv_cuda_devmem( float *v1, float *v2, float *vr, int len )
{
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = len;
    //size_t size = numElements * sizeof(float);
    //printf("[Vector addition of %d elements]\n", numElements);


    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("vdiv:CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorDiv<<<blocksPerGrid, threadsPerBlock>>>(v1, v2, vr, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorDiv kernel (error code %x:%s)!\n", err, hipGetErrorString(err));
	return -1;
    }

//    printf("Done\n");
    return 0;
}
