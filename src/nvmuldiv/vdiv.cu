
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>


__global__ void
vectorDiv(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements) {
		C[i] = A[i] / B[i];
	}
}

int vdiv_cuda( float v1[], float v2[], float vr[], int len )
{
	
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = len;
    size_t size = numElements * sizeof(float);
//    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
//    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, v1, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, v2, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 1024;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("vdiv:CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorDiv<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorDiv kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
//    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(vr, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

#ifdef __NVMULDIV_ENABLE_VERIFY__
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(v1[i] / v2[i] - vr[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
	    fprintf(stderr, "v1[i]: %f v2[i]: %f vr[i]:%f, expected:%f\n", v1[i], v2[i], vr[i], v1[i]/v2[i]);
        }
    }
#endif

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

//    printf("Done\n");
    return 0;
}
