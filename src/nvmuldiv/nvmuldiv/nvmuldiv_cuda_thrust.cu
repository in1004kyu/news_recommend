#include "hip/hip_runtime.h"
#include <thrust/version.h>
#include <iostream>
#include <thrust/device_vector.h>

#include <stdlib.h>
#include <time.h>

#define NUM_MAX_DATA	1024
#define NUM_MIN_DATA	2

float reduce_mul_vector( thrust::device_vector<float> &d1 )
{
	return thrust::reduce(d1.begin(), d1.end(), (float) 1, thrust::multiplies<float>());
}

extern "C" float nvmuldiv_cuda_thrust( float numerators[], float denominators[], int len)
{
	std::vector<float> vnum (numerators, numerators + len );
	std::vector<float> vden (denominators, denominators + len );

	thrust::device_vector<float> d1( vnum );
	thrust::device_vector<float> d2( vden );
	float num = reduce_mul_vector( d1 );
	float den = reduce_mul_vector( d2 );
	return num / den;
}

/* --- CUDA - NO THRUST CALLS --- */
extern "C" int vdiv_cuda( float v1[], float v2[], float vr[], int len );

extern "C" float nvmuldiv_cuda( float numerators[], float denominators[], int len)
{
	float *vr = (float *) malloc( sizeof(float) * len );
	vdiv_cuda( numerators, denominators, vr, len );
	std::vector<float> vvr (vr, vr + len );

	thrust::device_vector<float> dvr( vvr );
	float score = reduce_mul_vector( dvr );

	free(vr);
	return score;
}

__global__ void
vectorDiv(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements) {
		C[i] = A[i] / B[i];
	}
}

extern "C" int vdiv_cuda( float v1[], float v2[], float vr[], int len )
{
	
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = len;
    size_t size = numElements * sizeof(float);
//    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
//    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, v1, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, v2, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
//    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorDiv<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
//    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(vr, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

#ifdef __NVMULDIV_ENABLE_VERIFY__
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(v1[i] / v2[i] - vr[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
	    fprintf(stderr, "v1[i]: %f v2[i]: %f vr[i]:%f, expected:%f\n", v1[i], v2[i], vr[i], v1[i]/v2[i]);
            exit(EXIT_FAILURE);
        }
    }
#endif

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Reset the device and exit
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

//    printf("Done\n");
    return 0;
}
