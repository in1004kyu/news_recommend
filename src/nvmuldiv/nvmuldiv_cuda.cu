#include <thrust/version.h>
#include <iostream>
#include <thrust/device_vector.h>
#include <stdio.h>

#include <stdlib.h>
#include <time.h>

#include "nvmuldiv.h"

extern "C" nvmuldiv_status_t _nvmuldiv_cuda_init(void)
{
	static bool deviceMapHostSet = false;
	hipError_t result = hipSuccess;
	if ( !deviceMapHostSet ) {
		result = hipSetDeviceFlags(hipDeviceMapHost);
		if ( result == hipSuccess ) {
			deviceMapHostSet = true;
		}
	}
	if ( result != hipSuccess ) {
        	fprintf(stderr, "_nvmuldiv_cuda_init(error code %x:%s)!\n", result, hipGetErrorString(result));
		return nvmuldiv_status_generic_cuda;
	}

	return nvmuldiv_status_success;
}

extern "C" float nvmuldiv_cuda( float numerators[], float denominators[], int len)
{
	float *vr = (float *) malloc( sizeof(float) * len );

	// divide: vr = n / d
	vdiv_cuda( numerators, denominators, vr, len );

	// multiply: reduce(vr)
	/*
	thrust::device_vector<float> dvr( vr, vr + len );
	float score = thrust::reduce(dvr.begin(), dvr.end(), (float) 1, thrust::multiplies<float>());
	*/
	float score = 0;
	reduce_mul( vr, len, &score);

	free(vr);
	return score;
}


extern "C" void * nvmuldiv_mapped_alloc( size_t size )
{
	void *p_h = NULL;
	hipError_t result;

	result = hipHostAlloc( &p_h, size, hipHostMallocMapped );
	if ( result != hipSuccess ) {
        	fprintf(stderr, "nvmuldiv_mapped_alloc(error code %x:%s)!\n", result, hipGetErrorString(result));
	}
	return p_h;
}

extern "C" void nvmuldiv_mapped_free( void *p_h )
{
	hipHostFree( p_h );
}

extern "C" void *nvmuldiv_mapped_device( void *p_h )
{
	void *p_d = NULL;
	hipError_t err;
	err = hipHostGetDevicePointer( &p_d, p_h, 0);
	if ( err != hipSuccess ) {
        	fprintf(stderr, "nvmuldiv_mapped_device(error code %x:%s)!\n", err, hipGetErrorString(err));
		return NULL;
	}
	return p_d;
}

extern "C" void *nvmuldiv_device_alloc( size_t size )
{
	void *p_d = NULL;
	hipError_t err = hipSuccess;
	err = hipMalloc( &p_d, size );
	if ( err != hipSuccess ) {
        	fprintf(stderr, "nvmuldiv_device_alloc(error code %x:%s)!\n", err, hipGetErrorString(err));
		return NULL;
	} else {
        	fprintf(stderr, "nvmuldiv_device_alloc: %p size:%ld\n", p_d, size);
	}

	return p_d;
}

extern "C" void nvmuldiv_device_free( void *p_d )
{
	hipFree( p_d );
}

extern "C" int nvmuldiv_seg_cuda_devmem( float *n_d, float *d_d, int len, float *r_d, int len_r, float *vr_d )
{
	int result = 0;
	bool localAlloc = false;
// alloc
	if ( vr_d == NULL ) {
		vr_d = (float *) nvmuldiv_device_alloc( len * sizeof(float) );
		localAlloc = true;
		//fprintf( stderr, "nvmuldiv_seg_cuda_devmem: vr_d: %p local-allocated\n", vr_d );
	}
// kernel
	/*result = vdiv_cuda_devmem( n_d, d_d, vr_d, len );
	if ( result == 0 ) {
		result = reduce_mul_seg_devmem( vr_d, len, r_d, len_r );
	}*/
	reduce_divmul_seg_devmem(n_d,d_d,vr_d,len,r_d,len_r);
// free
	if ( localAlloc ) {
		//fprintf( stderr, "nvmuldiv_seg_cuda_devmem: freeing vr_d: %p local-allocated\n", vr_d );
		nvmuldiv_device_free( vr_d );
	}
	return result;
}
void reduce_mul_dummy(float *v, int len, float *result )
{
	float r = 1;
	for ( int i = 0; i < len; i++ ) r *= v[i];
	*result = r;
}
extern "C" int nvmuldiv_seg_cuda( float n[], float d[], int len, float r[], int len_r )
{
	int result = -1;
#if 1
	float *vr = (float *) malloc( sizeof(float) * len );
	vdiv_cuda( n, d, vr, len );
	reduce_mul_seg( vr, len, r, len_r );
	/*
	for( int i = 0; i < len / len_r; i++ ) {
		reduce_mul( &vr[i*len_r], len_r, &r[i] );
		//reduce_mul_dummy( &vr[i*len_r], len_r, &r[i] );
	}
	*/
	free(vr);
	result = 0;
#else
	

	float *n_h = (float *) nvmuldiv_mapped_alloc( sizeof(float) * len );
	float *d_h = (float *) nvmuldiv_mapped_alloc( sizeof(float) * len );
	float *n_d;
	float *d_d;
	float *vr_d = NULL;
	float *r_d = NULL;
	hipError_t err;
	hipDeviceProp_t prop;
	hipGetDeviceProperties( &prop, 0 );
	long ts1, ts2, ts3, ts4;

	ts1 = nvmuldiv_timestamp();

	if ( !prop.canMapHostMemory) {
		result = -2;
		fprintf( stderr, "nvmuldiv: cannot map host memory to device\n" );
		return result;
	}

	err = hipMalloc( &vr_d, sizeof(float) * len );
	if ( err != hipSuccess ) {
		return -1;	// insufficient memory
	}
	if ( err == hipSuccess ) {
		err = hipMalloc( &r_d, sizeof(float) * (len / len_r) );
	}
	if ( err == hipSuccess ) {
		memcpy( n_h, n, len * sizeof(float) );
		memcpy( d_h, d, len * sizeof(float) );
	}

	if ( err == hipSuccess ) {
		err = hipHostGetDevicePointer( &n_d, n_h, 0);
	}
	if ( err == hipSuccess ) {
		err = hipHostGetDevicePointer( &d_d, d_h, 0);
	}
#if 0
	// vdiv
	//	n_d: input, page locked
	//	d_d: input, page locked
	// 	vr_d: output, intermediate, device memory
	if ( err == hipSuccess ) {
		fprintf(stderr, "nvmuldiv: n_h=%p, n_d=%p\n", n_h, n_d );
		result = vdiv_cuda_devmem( n_d, d_d, vr_d, len );
	}
	// reduce
	//	vr_d: input, device memory
	//	r_d: output, device_memory
	if ( err == hipSuccess && result == 0 ) {
		result = reduce_mul_seg_devmem( vr_d, len, r_d, len_r );
	}
#endif
	ts2 = nvmuldiv_timestamp();

	if ( err == hipSuccess ) {
		long ts1, ts2;
		ts1 = nvmuldiv_timestamp();
		result = nvmuldiv_seg_cuda_devmem( n_d, d_d, len, r_d, len_r, vr_d );
		ts2 = nvmuldiv_timestamp();
		fprintf( stderr, "nvmuldiv_seg_cuda_devmem(): %ldus\n", (ts2-ts1) );
	}
	ts3 = nvmuldiv_timestamp();
	// grab the result
	//	r_d: device memory
	//	r: host memory
	if ( err == hipSuccess && result == 0 ) {
		err = hipMemcpy( r_d, r, sizeof(float) * (len / len_r), hipMemcpyHostToDevice );
	}
	if ( vr_d != NULL ) hipFree( vr_d );
	if ( r_d != NULL ) hipFree( r_d );
	nvmuldiv_mapped_free( n_h );
	nvmuldiv_mapped_free( d_h );

	ts4 = nvmuldiv_timestamp();
	fprintf( stderr, "nvmuldiv_seg_cudam(): alloc/copy:%ldus, copy/free:%ldus\n", (ts2-ts1), (ts4-ts3));
#endif
	return result;
}
