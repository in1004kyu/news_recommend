#include "hip/hip_runtime.h"
#include <thrust/version.h>
#include <iostream>
#include <thrust/device_vector.h>

#include <stdlib.h>
#include <time.h>
#include <stdio.h>

#include "nvmuldiv.h"
#define MAX_BLOCKS		65535
#define MAX_THREADS_PER_BLOCK	1024

/* --- reduce --- */
__global__ void kern_reduce_mul(float *g_idata, float *g_odata) { 
	extern __shared__ float sdata[];

	// each thread loads one element from global to shared mem
	unsigned int tid = threadIdx.x;					// 0..127
	unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x; 	// 0..127
	sdata[tid] = g_idata[i] * g_idata[i+blockDim.x];		// 0..127, 0..127, 128..255
	__syncthreads();

	// do reduction in shared mem: 64 > 32 > 16 > 8 > 4> 2
	for(unsigned int s=blockDim.x/2; s > 32; s >>= 1) { 
		if (tid < s) {
			sdata[tid] *= sdata[tid + s]; 			// 0..63, 64...127
		}
		__syncthreads(); 
	}
	if ( tid < 32 ) {
		// unroll the loop at the last warp
		// within a warp where instructions are SIMD synchronous
		// no need __syncthreads
		sdata[tid] *= sdata[tid + 32];
		sdata[tid] *= sdata[tid + 16];
		sdata[tid] *= sdata[tid + 8];
		sdata[tid] *= sdata[tid + 4];
		sdata[tid] *= sdata[tid + 2];
		sdata[tid] *= sdata[tid + 1];
	}
	if ( tid == 0 ) {
		g_odata[blockIdx.x] = sdata[0];
	}
}

extern "C" int reduce_mul( float *v, int len, float *result )
{
	hipError_t err = hipSuccess;

	int dimBlock = 128;	// threads per block : half the length
	int dimGrid = ((len/2) + dimBlock - 1) / dimBlock;	// number of blocks
	int padded_len = dimGrid * (dimBlock * 2);	// threads x 2 x blocks
	size_t smemSize = padded_len/2 * sizeof(float);

	float *d_v = NULL;
	float *d_r = NULL;

	// Device Memory for input:d_v[padded_len] and output:d_r[dimGrid]
	err = hipMalloc( (void **) &d_v, padded_len * sizeof(float));
	if ( err == hipSuccess ) {
		err = hipMalloc( (void **) &d_r, dimGrid * sizeof(float));
	}

	if ( err == hipSuccess ) {
		err = hipMemcpy( d_v, v, len * sizeof(float), hipMemcpyHostToDevice );
	}

	// Launch Kernel: number of blocks, threads per block, shared memory size in bytes
	// printf( "reduce_mul: blocks:%d, threads per block:%d, smem size:%d\n", dimGrid, dimBlock, (int) smemSize);
	if ( err == hipSuccess ) {
		kern_reduce_mul<<< dimGrid, dimBlock, smemSize >>>(d_v, d_r);
		err = hipGetLastError();
	}

	if ( err == hipSuccess ) {
		float value = 1;
		float *r = (float *) malloc( sizeof(float) * dimGrid );
		err = hipMemcpy( r, d_r, sizeof(float) * dimGrid, hipMemcpyDeviceToHost);
		if ( err == hipSuccess ) {
			//printf( "d_r[] = {" );
			for ( int i = 0; i < dimGrid; i++ ) {
			//	printf( "%f,", r[i] );
				value *= r[i];
			}
			//printf( "}\n" );
			*result = value;
		}
	}

	if ( err == hipSuccess ) {
		err = hipFree(d_v);
	}
	if ( err == hipSuccess ) {
		hipFree(d_r);
	}
	if ( err != hipSuccess ) {
        	fprintf(stderr, "CUDA (error code: %x: %s)\n", err, hipGetErrorString(err));
	}

	return 0;
}

int aligned_length( int len )
{
	// 1024, 512, 256, 128, 64, 32, 16, 8, 4, 2
	int m = 4;
	while ( m < 1024 ) {
		if ( m >= len ) {
			break;
		}
		m <<= 1;
	}
	//printf( "aligned_length: len:%d -> m:%d\n", len, m );
	return m;
}
extern "C" int reduce_mul_seg( float v[], int len, float r[], int len_r )
{
	hipError_t err = hipSuccess;

	int numSegs = len / len_r;
	if ( numSegs > MAX_BLOCKS ) {
		fprintf( stderr, "reduce_mul_seg: too many segments (%d / %d = %d > MAX_BLOCKS (%d) )\n", len, len_r, len / len_r, MAX_BLOCKS );
		return -1;
	}
	int aligned_len_r = aligned_length( len_r );
	int dimBlock = aligned_len_r / 2;	// threads per block : half the length
	int dimGrid = numSegs;	// number of blocks: number of segments
	int padded_len = dimGrid * aligned_len_r;	// segment length x blocks
	size_t smemSize = aligned_len_r/2 * sizeof(float);

	float *d_v = NULL;
	float *d_r = NULL;


	// Device Memory for input:d_v[padded_len] and output:d_r[numSegs]
	err = hipMalloc( (void **) &d_v, padded_len * sizeof(float));
	if ( err == hipSuccess ) {
		err = hipMalloc( (void **) &d_r, numSegs * sizeof(float));
	}

	if ( err == hipSuccess ) {
		err = hipMemcpy( d_v, v, len * sizeof(float), hipMemcpyHostToDevice );
	}

	// Launch Kernel: number of blocks, threads per block, shared memory size in bytes
	// printf( "reduce_mul_seg: blocks:%d, threads per block:%d, smem size:%d\n", dimGrid, dimBlock, (int) smemSize);
	if ( err == hipSuccess ) {
		kern_reduce_mul<<< dimGrid, dimBlock, smemSize >>>(d_v, d_r);
		err = hipGetLastError();
	}

	if ( err == hipSuccess ) {
		err = hipMemcpy( r, d_r, sizeof(float) * numSegs, hipMemcpyDeviceToHost);
#ifdef __NVMULDIV_ENABLE_VERIFY__
		if ( err == hipSuccess ) {
			printf( "d_r[] = {" );
			for ( int i = 0; i < dimGrid; i++ ) {
				printf( "%f,", r[i] );
			}
			printf( "}\n" );
		}
#endif
	}

	if ( err == hipSuccess ) {
		err = hipFree(d_v);
	}
	if ( err == hipSuccess ) {
		hipFree(d_r);
	}
	if ( err != hipSuccess ) {
        	fprintf(stderr, "CUDA (error code: %x: %s)\n", err, hipGetErrorString(err));
	}

	return 0;
}
